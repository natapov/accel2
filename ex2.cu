#include "hip/hip_runtime.h"
#include "ex2.h"
#include <cuda/atomic>

__device__
void process_image(uchar *target, uchar *reference, uchar *result) {
    // TODO complete according to hw1
}

__global__
void process_image_kernel(uchar *target, uchar *reference, uchar *result){
    process_image(target, reference, result);
}

class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)

public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
    }

    bool enqueue(int job_id, uchar *target, uchar *reference, uchar *result) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
        return false;
    }

    bool dequeue(int *job_id) override
    {
        return false;

        // TODO query (don't block) streams for any completed requests.
        //for ()
        //{
            hipError_t status = hipStreamQuery(0); // TODO query diffrent stream each iteration
            switch (status) {
            case hipSuccess:
                // TODO return the img_id of the request that was completed.
                //*img_id = ...
                return true;
            case hipErrorNotReady:
                return false;
            default:
                CUDA_CHECK(status);
                return false;
            }
        //}
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

// TODO implement a SPSC queue
// TODO implement the persistent kernel
// TODO implement a function for calculating the threadblocks count

class queue_server : public image_processing_server
{
private:
    // TODO define queue server context (memory buffers, etc...)
public:
    queue_server(int threads)
    {
        // TODO initialize host state
        // TODO launch GPU persistent kernel with given number of threads, and calculated number of threadblocks
    }

    ~queue_server() override
    {
        // TODO free resources allocated in constructor
    }

    bool enqueue(int job_id, uchar *target, uchar *reference, uchar *result) override
    {
        // TODO push new task into queue if possible
        return false;
    }

    bool dequeue(int *job_id) override
    {
        // TODO query (don't block) the producer-consumer queue for any responses.
        return false;

        // TODO return the job_id of the request that was completed.
        //*job_id = ... 
        return true;
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
