#include "hip/hip_runtime.h"
#include "ex2.h"
#include <cuda/atomic>
using namespace std;

#define NUM_STREAMS 64

// Our functions from ex 1
const int img_size = SIZE * SIZE * CHANNELS;
__device__ void prefixSum(int arr[], int size, int tid, int threads) {
    int increment;
    const auto is_active = tid < size;
    for (int stride = 1; stride<size; stride*=2) {
        if (tid >= stride && is_active) {
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && is_active) {
            arr[tid] += increment;
        }
        __syncthreads();
    }
}

__device__ void argmin(int arr[], int len, int tid, int threads) {
    int halfLen = len / 2;
    bool firstIteration = true;
    int prevHalfLength = 0;
    while (halfLen > 0) {
        if(tid < halfLen){
            if(arr[tid] == arr[tid + halfLen]){ //a corenr case
                int lhsIdx = tid;
                int rhdIdx = tid + halfLen;
                int lhsOriginalIdx = firstIteration ? lhsIdx : arr[prevHalfLength + lhsIdx];
                int rhsOriginalIdx = firstIteration ? rhdIdx : arr[prevHalfLength + rhdIdx];
                arr[tid + halfLen] = lhsOriginalIdx < rhsOriginalIdx ? lhsOriginalIdx : rhsOriginalIdx;
            }
            else { //the common case
                bool isLhsSmaller = (arr[tid] < arr[tid + halfLen]);
                int idxOfSmaller = isLhsSmaller * tid + (!isLhsSmaller) * (tid + halfLen);
                int smallerValue = arr[idxOfSmaller];
                int origIdxOfSmaller = firstIteration * idxOfSmaller + (!firstIteration) * arr[prevHalfLength + idxOfSmaller];
                arr[tid] = smallerValue;
                arr[tid + halfLen] = origIdxOfSmaller;
            }
        }
        __syncthreads();
        firstIteration = false;
        prevHalfLength = halfLen;
        halfLen /= 2;
    }
}

__device__ void zero_array(int* histograms, int size=CHANNELS*LEVELS) {
    auto hist_flat = (int*) histograms;
    const int tid = threadIdx.x;
    const int threads = blockDim.x;
    for(int i = tid; i < size; i+=threads) {
        hist_flat[i] = 0;
    }
}

__device__ void colorHist(uchar img[][CHANNELS], int histograms[][LEVELS]) {
    const int pic_size = SIZE * SIZE;
    const int tid = threadIdx.x;
    const int threads = blockDim.x;

    for (int i = tid; i < 3*pic_size; i+=threads) {
        const int color = i%3;
        const int pixel = i/3;
        assert(pixel < pic_size);
        atomicAdd(&histograms[color][img[pixel][color]], 1);
    }
}


__device__ void performMapping(int maps[][LEVELS], uchar targetImg[][CHANNELS], uchar resultImg[][CHANNELS]){
    int pixels = SIZE * SIZE;
    const int tid = threadIdx.x;
    const int threads = blockDim.x;
    for (int i = tid; i < pixels; i+= threads) {
        uchar *inRgbPixel = targetImg[i];
        uchar *outRgbPixel = resultImg[i];
        for (int j = 0; j < CHANNELS; j++){
            int *mapChannel = maps[j];
            outRgbPixel[j] = mapChannel[inRgbPixel[j]];
        }
    }    
}
// Our functions from ex 1 end



__device__ void process_image(uchar *targets, uchar *references, uchar *results,int deleta_cdf_row[LEVELS], int map_cdf[][LEVELS], int histogramsShared_target[][LEVELS], int histogramsShared_refrence[][LEVELS]) {
    int tid = threadIdx.x;;
    int threads = blockDim.x;
    int bid = blockIdx.x;
    zero_array((int*)histogramsShared_target,   CHANNELS * LEVELS);
    zero_array((int*)histogramsShared_refrence, CHANNELS * LEVELS);
    zero_array((int*)map_cdf,                   CHANNELS * LEVELS);
    zero_array((int*)deleta_cdf_row,            LEVELS);

    auto target   = (uchar(*)[CHANNELS]) &targets[  bid * img_size];
    auto refrence = (uchar(*)[CHANNELS]) &references[bid * img_size];
    auto result   = (uchar(*)[CHANNELS]) &results[  bid * img_size];

    colorHist(target, histogramsShared_target);
    colorHist(refrence, histogramsShared_refrence);
    __syncthreads();

    for(int c=0; c < CHANNELS; c++)
    {   
        prefixSum(histogramsShared_target[c],LEVELS, threadIdx.x, blockDim.x);
        prefixSum(histogramsShared_refrence[c], LEVELS, threadIdx.x, blockDim.x);
        __syncthreads();

        for (int i = 0; i < LEVELS; i+=1) {
            for (int j = tid; j < LEVELS; j+=threads) {
                deleta_cdf_row[j] = abs(histogramsShared_target[c][i]-histogramsShared_refrence[c][j]);
            }
            __syncthreads();
            argmin(deleta_cdf_row, LEVELS, threadIdx.x, blockDim.x);
            __syncthreads();

            map_cdf[c][i] = deleta_cdf_row[1];

            __syncthreads();
        }
        __syncthreads();
    }          

    //Preform Map
    performMapping(map_cdf, target, result); 
    __syncthreads(); 
}

__global__
void process_image_kernel(uchar *targets, uchar *references, uchar *results){
    __shared__ int deleta_cdf_row[LEVELS];
    __shared__ int map_cdf[CHANNELS][LEVELS];
    __shared__ int histogramsShared_target[CHANNELS][LEVELS];
    __shared__ int histogramsShared_refrence[CHANNELS][LEVELS];

    process_image(targets, references, results, deleta_cdf_row, map_cdf, histogramsShared_target, histogramsShared_refrence);
}

class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
    int current_stream;
    hipStream_t streams[NUM_STREAMS];


public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
        current_stream=0;
        

        for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);

        }
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
        
        for (int i = 0; i < NUM_STREAMS; i++) {
            hipStreamDestroy(streams[i]);
        }
    }

    bool enqueue(int job_id, uchar *target, uchar *reference, uchar *result) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
        return false;
    }

    bool dequeue(int *job_id) override
    {
        return false;

        // TODO query (don't block) streams for any completed requests.
        //for ()
        //{
            hipError_t status = hipStreamQuery(0); // TODO query diffrent stream each iteration
            switch (status) {
            case hipSuccess:
                // TODO return the img_id of the request that was completed.
                //*img_id = ...
                return true;
            case hipErrorNotReady:
                return false;
            default:
                CUDA_CHECK(status);
                return false;
            }
        //}
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

class Que
{
private:
    // On each use of the flag, we switch the meaning of its values (true/false -> data is ready/not-ready)

    cuda::atomic<bool> lock;
    
    static const int que_max = 16;
    Job que[que_max];
    int front_of_que = 0;
    int size = 0;

public:
    Que() : lock(false){
    }

    __device__ __host__ bool dequeue(Job* job)
    {
        bool success = false;
        while (lock.exchange(true ,cuda::memory_order_relaxed) == false);
        cuda::atomic_thread_fence(cuda::memory_order_acquire, cuda::thread_scope_system);
        if(size) {
            *job = que[front_of_que%que_max];
            size -= 1;
            front_of_que += 1;
            success = true;
        }
        lock.store(false, cuda::memory_order_release);
        return success;
    }

    __device__ __host__ bool enqueue(Job& job) {
        while (lock.exchange(true ,cuda::memory_order_relaxed) == false);
        cuda::atomic_thread_fence(cuda::memory_order_acquire, cuda::thread_scope_system);

        bool success = false;
        if(size < que_max) {
            que[(front_of_que + size)%que_max] = job;
            // que[front_of_que + size] = {job_id, target, reference, result};

            size += 1;
            success = true;
        }
        lock.store(false, cuda::memory_order_release);
        return success;
    }

    __device__ __host__ bool enqueue(int job_id, uchar* target, uchar* reference, uchar* result) {
        while (lock.exchange(true ,cuda::memory_order_relaxed) == false);
        cuda::atomic_thread_fence(cuda::memory_order_acquire, cuda::thread_scope_system);

        bool success = false;
        if(size < que_max) {
            que[(front_of_que + size)%que_max] = {job_id, target, reference, result};
            // que[front_of_que + size] = {job_id, target, reference, result};

            size += 1;
            success = true;
        }
        lock.store(false, cuda::memory_order_release);
        return success;
    }
};

__global__ void kernel(Que* input, Que* output)
{
    __shared__ int deleta_cdf_row[LEVELS];
    __shared__ int map_cdf[CHANNELS][LEVELS];
    __shared__ int histogramsShared_target[CHANNELS][LEVELS];
    __shared__ int histogramsShared_refrence[CHANNELS][LEVELS];
    
    Job job;
    while(true) {
        if(input->dequeue(&job)) {
            process_image(job.target, job.reference, job.result, deleta_cdf_row, map_cdf, histogramsShared_target, histogramsShared_refrence);
            output->enqueue(job);
        }
    }
}


// TODO implement a SPSC queue
// TODO implement a function for calculating the threadblocks count

typedef struct Job {
    int job_id;
    uchar* target;
    uchar* reference;
    uchar* result;
}Job;

class queue_server : public image_processing_server
{
private:
    void* pinned_host_buffer;
    Que* que_host_to_gpu;
    Que* que_gpu_to_host;
public:
    // Job& operator[](int index) {
    //     return que[index % que_max];
    // }


    queue_server(int threads) {

        // Allocate pinned host buffer for two shared_memory instances
        hipHostMalloc(&pinned_host_buffer, 2 * sizeof(Que));
        // Use placement new operator to construct our class on the pinned buffer
        que_host_to_gpu = new (pinned_host_buffer) Que();
        que_gpu_to_host = new (pinned_host_buffer + sizeof(Que)) Que();
        kernel<<<threads, 1>>>(que_host_to_gpu, que_gpu_to_host);
        // TODO initialize host state
        // TODO launch GPU persistent kernel with given number of threads, and calculated number of threadblocks
    }

    ~queue_server() override {
        que_host_to_gpu->~Que();
        que_gpu_to_host->~Que();
        auto err = hipHostFree(pinned_host_buffer);
        assert(err == hipSuccess);
        hipDeviceReset();
    }

    bool enqueue(int job_id, uchar *target, uchar *reference, uchar *result) override
    {
        return que_host_to_gpu->enqueue(job_id, target, reference, result);
    }

    bool dequeue(int *job_id) override
    {
        Job* job;
        bool success = false;
        if(que_gpu_to_host->dequeue(job)) {
            *job_id = job->job_id;
            success= true;
        }
        return success;
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
